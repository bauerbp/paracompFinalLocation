#include "hip/hip_runtime.h"
#include "../paracompFinalConstants/constants.cu"
#include <math.h>

//numFFTPairs is number of mics which have a set of matches
__global__
void findCoordinate(int numFFTPairs, int numMics, Coordinate* micCoordinates,
                    WavePairContainer* filteredPairs) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= numFFTPairs){
    return;
  }

  if (tid==0) int referenceMic=filteredPairs[0].firstFFT;
  
  for(int i=1; i<=2;i++){
    if(tid+i<numFFTPairs){
      int mic1 = filteredPairs[tid].firstFFT;
      int mic2 = filteredPairs[tid].secondFFT;
      decimal pairDistance = sqrt(pow(micCoordinates[mic2].x - micCoordinats[mic1].x ,2)+pow(micCoordinates[mic2].y-micCoordinates[mic1].y,2));
      //phase^2 = (x-x1)^2 + (y-y1)^2
      if(filteredPairs[tid].firstFFT.offset + filteredPairs[tid].secondFFT.offset < pairDistance || //if dist > r1+r2
         fabs(filteredPairs[tid].firstFFT.offset - filteredPairs[tid].secondFFT.offset) > pairDistance || //if dist < |r1-r2|
         (pairDistance==0 && filteredPairs[tid].firstFFT.offset==filteredPairs[tid].secondFFT.offset)) //if (dist==0 && r1==r2)
         return;
      
             

  
  /*
  decimal micDistances[numFFTPairs];

  micDistances[tid]=0;
  referenceMic = filteredPairs[0].firstFFT;
  int mic1 = filteredPairs[idx].firstFFT;
  int mic2 = filteredPairs[idx].secondFFT;
  if (mic1 == referenceMic || mic2 == referenceMic){
     tempDistances = sqrt(micCoordinates[mic1].x * micCoordinates[mic1].x + micCoordinates[mic2].y * micCoordinates[mic2].y);

    (mic1 != referenceMic) ? micDistances[mic1]=tempDistance : micDistances[mic2]=tempDistance;
  }*/
  //speed of sound is about 34320 centimeters per second (ma.
  
  //go through list of matches of first mic, and take the join of the matches of the rest of the mics.

  
