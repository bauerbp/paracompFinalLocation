#include "hip/hip_runtime.h"
#include "../paracompFinalConstants/constants.cu"
#include <math.h>

__global__
void findCoordinate(int numFFTPairs, Coordinate* micCoordinates,
                    WavePairContainer* filteredPairs) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= numPairs){
    return;
  }

  decimal micDistances[numPairs];

  int mic1 = filteredPairs[idx].firstFFT;
  int mic2 = filteredPairs[idx].secondFFT;
  micDistances[tid] = sqrt(micCoordinates[mic1].x * micCoordinates[mic1].x +
                      micCoordinates[mic2].y * micCoordinates[mic2].y);
  
  //speed of sound is about 34320 centimeters per second (ma.
  
